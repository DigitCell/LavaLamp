#include "hip/hip_runtime.h"
#include "cuda_solver.cuh"

__constant__  gpuParams params;

__device__ int GetIndex(int x, int y)
{
    return y*params.mapWidth+x;
}
__device__ int GetIndexMap(int i,int x, int y)
{
    return  i*params.maxMapCoeff1+y*params.mapWidth+x;
}

__device__ int GetIndexNeightb(int i, int j)
{
    return  i*maxSprings+j;
}

Cuda_solver::Cuda_solver()
{

         CudaInit();
        // params init

        h_params=new gpuParams;

        h_params->yield=0.08f;
        h_params->stiffness= 0.18f;
        h_params->nearStiffness= 0.01f;
        h_params->linearViscocity =0.5f;
        h_params->quadraticViscocity= 1.f;

        h_params->temp_increase=0.05f;
        h_params->temp_decrease=0.005f;
        h_params->gravity_coeff_max=11.0f;
        h_params->gravity_coeff_min=3.0f;

        h_params->totalParticles=0;

        gravity.x=0.0f;
        gravity.y=-9.8f;
        h_params->gravity.x=gravity.x;
        h_params->gravity.y=gravity.y;

        h_params->maxMapCoeff1=mapWidth*mapHeight;
        h_params->maxMapCoeff2=mapWidth*mapHeight;

        h_params->mapWidth=mapWidth;
        h_params->mapHeight=mapHeight;


        h_particles=(Particle*)malloc(sizeof(Particle)*maxParticles);
        //h_neighbours=(Springs*)malloc(sizeof(Springs)*maxParticles);
        h_prevPos=(float2*)malloc(sizeof(float2)*maxParticles);

        h_particleTypes=(ParticleType*)malloc(sizeof(ParticleType)*maxParticles);
        h_savedParticleColors=(Vec4*)malloc(sizeof(Vec4)*maxParticles);



        h_map=(int*)malloc(sizeof(int)*mapWidth*mapHeight*maxParticles);
        h_map_size=(int*)malloc(sizeof(int)*mapWidth*mapHeight);

        h_mapCoords=(int2*)malloc(sizeof(int2)*maxParticles);
        h_boundaries=(float3*)malloc(sizeof(float3)*4);

        h_neightb_size=(int*)malloc(sizeof(int)*maxParticles);


        checkCudaCall(hipMalloc(&d_particles,sizeof(Particle)*maxParticles));

        checkCudaCall(hipMalloc(&d_prevPos,sizeof(float2)*maxParticles));
        checkCudaCall(hipMalloc(&d_particleTypes,sizeof(ParticleType)*maxParticles));
        checkCudaCall(hipMalloc(&d_savedParticleColors,sizeof(Vec4)*maxParticles));


        checkCudaCall(hipMalloc(&d_map,sizeof(int)*mapWidth*mapHeight*maxParticles));
        checkCudaCall(hipMalloc(&d_map_size,sizeof(int)*mapWidth*mapHeight));

        checkCudaCall(hipMalloc(&d_mapCoords,sizeof(int2)*maxParticles));
        checkCudaCall(hipMalloc(&d_boundaries,sizeof(float3)*4));

        gravity=make_float2(0.f, -9.0f);

        for(int i=0;i<4;i++)
        {
            h_boundaries[i].x=boundaries[i].x;
            h_boundaries[i].y=boundaries[i].y;
            h_boundaries[i].z=boundaries[i].c;

        }

        checkCudaCall(hipMalloc(&d_neightb_index,sizeof(int)*maxSprings*maxParticles));
        checkCudaCall(hipMalloc(&d_neightb_size,sizeof(int)*maxParticles));
        checkCudaCall(hipMalloc(&d_neightb_r,sizeof(float)*maxSprings*maxParticles));
        checkCudaCall(hipMalloc(&d_neightb_Lij,sizeof(float)*maxSprings*maxParticles));



        memset(h_map, 0, mapWidth*mapHeight*sizeof(int));

        memset(h_neightb_size, 0, maxParticles* sizeof(int));

        checkCudaCall(hipMemcpyToSymbol(HIP_SYMBOL(params),h_params,sizeof(gpuParams)));
        UpdateGPUBuffers();
        ClearMap();
        updateMap();

}


void Cuda_solver::CudaInit()
{
    int devID = gpuGetMaxGflopsDeviceId();
    checkCudaErrors(hipSetDevice(devID));
    int major = 0, minor = 0;
    checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, devID));
    checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, devID));
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
           devID, _ConvertSMVer2ArchName(major, minor), major, minor);
}


void Cuda_solver::generateParticles(){
    if (h_params->totalParticles == maxParticles)
        return;

    if (delay++ < 2)
        return;

    for (int turn = 0; turn<2; turn++){
        Source& source = sources[turn];
        if (source.count >= maxParticles / 2) continue;

        for (int i = 0; i <= 2 && h_params->totalParticles<maxParticles; i++){
            Particle& p = h_particles[h_params->totalParticles];
            ParticleType& pt = h_particleTypes[h_params->totalParticles];
            h_params->totalParticles++;

            source.count++;

            //for an even distribution of particles
            float offset = float(i) / 1.5f;
            offset *= 0.2f;
            p.posX = source.position.x - offset*source.direction.y;
            p.posY = source.position.y + offset*source.direction.x;
            p.velX = source.speed *source.direction.x;
            p.velY = source.speed *source.direction.y;
            p.m = source.pt.mass;
            p.temp=0;

            pt = source.pt;
        }
    }
    delay = 0;

    UpdateGPUBuffers();
}

void Cuda_solver::UpdateGPUBuffers()
{
    //checkCudaCall(hipMemcpyToSymbol(HIP_SYMBOL(params),h_params,sizeof(gpuParams)));

    checkCudaCall(hipMemcpy(d_boundaries,h_boundaries,sizeof(float3)*4,hipMemcpyHostToDevice));

    checkCudaCall(hipMemcpy(d_particles,h_particles,sizeof(Particle)*maxParticles,hipMemcpyHostToDevice));

    checkCudaCall(hipMemcpy(d_neightb_size,h_neightb_size,sizeof(int)*maxParticles,hipMemcpyHostToDevice));
    //checkCudaCall(hipMemcpy(d_neighbours,h_neighbours,sizeof(Springs)*maxParticles,hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(d_prevPos,h_prevPos,sizeof(float2)*maxParticles,hipMemcpyHostToDevice));

    checkCudaCall(hipMemcpy(d_particleTypes,h_particleTypes,sizeof(ParticleType)*maxParticles,hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(d_savedParticleColors,h_savedParticleColors,sizeof(int4)*maxParticles,hipMemcpyHostToDevice));

    //checkCudaCall(hipMemcpy(d_map,h_map,sizeof(int)*mapWidth*mapHeight,hipMemcpyHostToDevice));
    //checkCudaCall(hipMemcpy(d_mapCoords,h_mapCoords,sizeof(int2)*mapWidth*mapHeight,hipMemcpyHostToDevice));

}

void Cuda_solver::UpdateHostBuffers()
{

    checkCudaCall(hipMemcpy(h_particles,d_particles,sizeof(Particle)*maxParticles,hipMemcpyDeviceToHost));
  //  checkCudaCall(hipMemcpy(h_neighbours,d_neighbours,sizeof(Springs)*maxParticles,hipMemcpyDeviceToHost));
  //  checkCudaCall(hipMemcpy(h_prevPos,d_prevPos,sizeof(float2)*maxParticles,hipMemcpyDeviceToHost));

  //  checkCudaCall(hipMemcpy(h_particleTypes,d_particleTypes,sizeof(ParticleType)*maxParticles,hipMemcpyDeviceToHost));
    checkCudaCall(hipMemcpy(h_savedParticleColors,d_savedParticleColors,sizeof(int4)*maxParticles,hipMemcpyDeviceToHost));
  //  checkCudaCall(hipMemcpy(h_map_size,d_map_size,sizeof(int)*mapWidth*mapHeight,hipMemcpyDeviceToHost));
  //  checkCudaCall(hipMemcpy(h_map,d_map,sizeof(int)*mapWidth*mapHeight*maxParticles,hipMemcpyDeviceToHost));
  //  checkCudaCall(hipMemcpy(h_mapCoords,d_mapCoords,sizeof(int*)*mapWidth*mapHeight,hipMemcpyDeviceToHost));

}



//2-Dimensional gravity for player input
void Cuda_solver::applyTemp(){

    if(h_params->totalParticles>1)
    {
        int threadsPerBlock = 256;
        int blocksPerGrid =
                (h_params->totalParticles + threadsPerBlock - 1) / threadsPerBlock;

        CudaApplyTemp<<<blocksPerGrid, threadsPerBlock>>>(d_particles);
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
    }
}

__global__ void CudaApplyTemp(Particle* particles)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i<params.totalParticles)
    {
        Particle& p = particles[i];
        if(p.posY<1.95 and p.posY>0.25)
            p.temp+=params.temp_increase*(1.0-p.posY/2.0);
        else
            p.temp-=params.temp_decrease;
        if(p.temp>params.gravity_coeff_max)
            p.temp=params.gravity_coeff_max;
        if(p.temp<params.gravity_coeff_min)
            p.temp=params.gravity_coeff_min;

    }
}

//2-Dimensional gravity for player input
void Cuda_solver::applyGravity()
{
     if(h_params->totalParticles>0)
    {
         int threadsPerBlock = 256;
         int blocksPerGrid =
                 (h_params->totalParticles + threadsPerBlock - 1) / threadsPerBlock;

        CudaApplyGravity<<<blocksPerGrid, threadsPerBlock>>>(d_particles);
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
    }
}

__global__ void CudaApplyGravity(Particle* particles)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i<params.totalParticles)
    {
        Particle& p = particles[i];

     //   if(i==0)
     //        printf("coord %f %f",p.posX, p.posY);

        p.velY += params.gravity.y*deltaTime+p.temp*deltaTime;
        p.velX += params.gravity.x*deltaTime;

    }
}

//applies viscosity impulses to particles
void Cuda_solver::applyViscosity()
{
     if(h_params->totalParticles>0)
    {
         int threadsPerBlock = 256;
         int blocksPerGrid =
                 (h_params->totalParticles + threadsPerBlock - 1) / threadsPerBlock;

        CudaApplyViscosity<<<blocksPerGrid, threadsPerBlock>>>(d_particles,
                                                               d_neightb_index, d_neightb_size, d_neightb_r, d_neightb_Lij);
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
    }
}

__global__ void CudaApplyViscosity(Particle* particles,
                                   int* neightb_index, int* neightb_size, float* neightb_r, float* neightb_Lij)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i<params.totalParticles)
    {
        Particle& p = particles[i];

        for (int j = 0; j < neightb_size[i]; j++){
            const Particle& pNear = particles[neightb_index[GetIndexNeightb(i,j)]];

            float diffX = pNear.posX - p.posX;
            float diffY = pNear.posY - p.posY;

            float r2 = diffX*diffX + diffY*diffY;
            float r = sqrt(r2);

            float q = r / particleHeight;

            if (q>1) continue;

            float diffVelX = p.velX - pNear.velX;
            float diffVelY = p.velY - pNear.velY;
            float u = diffVelX*diffX + diffVelY*diffY;

            if (u > 0){
                float a = 1 - q;
                diffX /= r;
                diffY /= r;
                u /= r;

                float I = 0.5f * deltaTime * a * (params.linearViscocity*u + params.quadraticViscocity*u*u);

                particles[i].velX -= I * diffX;
                particles[i].velY -= I * diffY;
            }
        }

    }
}

//Advances particle along its velocity
void Cuda_solver::advance()
{
     if(h_params->totalParticles>0)
    {
         int threadsPerBlock = 256;
         int blocksPerGrid =
                 (h_params->totalParticles + threadsPerBlock - 1) / threadsPerBlock;

        CudaAdvance<<<blocksPerGrid, threadsPerBlock>>>(d_particles, d_prevPos);
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
    }
}


__global__ void CudaAdvance(Particle* particles, float2* prevPos)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i<params.totalParticles)
    {
        Particle& p = particles[i];

        prevPos[i].x = p.posX;
        prevPos[i].y = p.posY;

        p.posX += deltaTime * p.velX;
        p.posY += deltaTime * p.velY;

    }
}


void Cuda_solver::adjustSprings()
{
     if(h_params->totalParticles>0)
    {
         int threadsPerBlock = 256;
         int blocksPerGrid =
                 (h_params->totalParticles + threadsPerBlock - 1) / threadsPerBlock;

        CudaAdjustSprings<<<blocksPerGrid, threadsPerBlock>>>(d_particles,
                                                              d_neightb_index, d_neightb_size, d_neightb_r, d_neightb_Lij);
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
    }
}

__global__ void CudaAdjustSprings(Particle* particles,
                                  int* neightb_index, int* neightb_size, float* neightb_r, float* neightb_Lij)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i<params.totalParticles)
    {
        const Particle& p = particles[i];
        //iterate through that particles neighbors
        for (int j = 0; j < neightb_size[i]; j++){
            const Particle& pNear =  particles[neightb_index[GetIndexNeightb(i,j)]];

            float r = neightb_r[GetIndexNeightb(i,j)];
            float q = r / particleHeight;

            if (q < 1 && q > 0.0000000000001f){
                float d = params.yield*neightb_Lij[GetIndexNeightb(i,j)];

                //calculate spring values
                if (r>particleHeight + d){
                    neightb_Lij[GetIndexNeightb(i,j)]+= deltaTime*alphaSpring*(r - particleHeight - d);
                }
                else if (r<particleHeight - d){
                    neightb_Lij[GetIndexNeightb(i,j)] -= deltaTime*alphaSpring*(particleHeight - d - r);
                }

                //apply those changes to the particle
                float Lij = neightb_Lij[GetIndexNeightb(i,j)];
                float diffX = pNear.posX - p.posX;
                float diffY = pNear.posY - p.posY;
                float displaceX = deltaTime*deltaTime*kSpring*(1 - Lij / particleHeight)*(Lij - r)*diffX;
                float displaceY = deltaTime*deltaTime*kSpring*(1 - Lij / particleHeight)*(Lij - r)*diffY;
                particles[i].posX -= 0.5f*displaceX;
                particles[i].posY -= 0.5f*displaceY;
            }
        }
    }
}



//Resets the map of the scene, re-adding every particle based on where it is at this moment
void Cuda_solver::updateMap()
{
    ClearMap();

    if(h_params->totalParticles>0)
    {
        int threadsPerBlock = 256;
        int blocksPerGrid =
                (h_params->totalParticles + threadsPerBlock - 1) / threadsPerBlock;


        CudaUpdateMap<<<blocksPerGrid, threadsPerBlock>>>(d_particles, d_map, d_map_size, d_mapCoords);
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
    }
}


__global__ void CudaUpdateMap(Particle* particles, int* map, int *map_size, int2* mapCoords)
{

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i<params.totalParticles)
    {
        Particle& p = particles[i];
        int x = p.posX / particleHeight;
        int y = p.posY / particleHeight;

        if (x < 1) x = 1;
        else if (x > params.mapWidth - 2) x = params.mapWidth - 2;

        if (y < 1)
            y = 1;
        else if (y > params.mapHeight - 2)
            y = params.mapHeight - 2;

        //this handles the linked list between particles on the same square

        int& indexAdd=map_size[GetIndex(x,y)];
/*
        if(indexAdd>0)
        {
           printf(" PrePart %i, mapsize %i , x=%i, y=%i" ,i, indexAdd,x,y );
        }
*/
        map[GetIndexMap(atomicAdd(&indexAdd,1),x,y)] = i;
       // int oldIndex =atomicAdd(&indexAdd,1);

        mapCoords[i].x = x;
        mapCoords[i].y = y;

    }
}


//Resets the map of the scene, re-adding every particle based on where it is at this moment
void Cuda_solver::ClearMap()
{

    int threadsPerBlock = 256;
    int blocksPerGrid =
            (h_params->maxMapCoeff1 + threadsPerBlock - 1) / threadsPerBlock;

    CudaClearMap<<<blocksPerGrid, threadsPerBlock>>>(d_map, d_map_size);
    hipDeviceSynchronize();
    getLastCudaError("Kernel execution failed");
}

__global__ void CudaClearMap(int* map, int *map_size)
{

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i<params.maxMapCoeff1)
    {
        map_size[i]=0;
        if(map_size[i]>0)
           printf("not Clear mapsize %i" , map_size[i] );
    }
}


//saves neighbors for lookup in other functions
void Cuda_solver::storeNeighbors(){
     if(h_params->totalParticles>0)
    {
         int threadsPerBlock = 256;
         int blocksPerGrid =
                 (h_params->totalParticles + threadsPerBlock - 1) / threadsPerBlock;

        CudaStoreNeighbors<<<blocksPerGrid, threadsPerBlock>>>(d_particles,  d_map, d_map_size, d_mapCoords,
                                                               d_neightb_index, d_neightb_size, d_neightb_r, d_neightb_Lij);
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
    }
}


__global__ void CudaStoreNeighbors(Particle* particles,  int* map, int *map_size, int2* mapCoords,
                                   int* neightb_index, int* neightb_size, float* neightb_r, float* neightb_Lij)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i<params.totalParticles)
    {
        Particle& p = particles[i];
        int pX = mapCoords[i].x;
        int pY = mapCoords[i].y;

        neightb_size[i]=0;

        //iterate over the nine squares on the grid around p
        for (int mapX = pX - 1; mapX <= pX + 1; mapX++){
            for (int mapY = pY - 1; mapY <= pY + 1; mapY++){
                //go through the current square's linked list of overlapping values, if there is one
               // for (Particle* nextDoor = &particles[map[GetIndex(mapX,mapY)]]; nextDoor != NULL; nextDoor = nextDoor->next){

                if(mapX<0 or mapY<0 or mapX>params.mapWidth-1 or mapY>params.mapHeight-1)
                     continue;
                if(map_size[GetIndex(mapX,mapY)]==0)
                    continue;
                for(int ip=0; ip<map_size[GetIndex(mapX,mapY)];ip++)
                {
                    const Particle& pj =  particles[map[GetIndexMap(ip,mapX,mapY)]];

                    float diffX = pj.posX - p.posX;
                    float diffY = pj.posY - p.posY;
                    float r2 = diffX*diffX + diffY*diffY;
                    float r = sqrt(r2);
                    float q = r / particleHeight;

                    //save this neighbor
                    if (q < 1 && q > 0.0000000000001f){

                        const int j=neightb_size[i];// (neightb_size[i]==0)?0:neightb_size[i]-1;
                        if (neightb_size[i] < maxSprings){
                            neightb_index[GetIndexNeightb(i, j)]=map[GetIndexMap(ip,mapX,mapY)];
                            neightb_r[GetIndexNeightb(i,j)]=r;
                            neightb_Lij[GetIndexNeightb(i,j)]=particleHeight;
                            neightb_size[i]++;
                        }

                    }
                }
            }
        }

    }
}


//This maps pretty closely to the outline in the paper. Find density and pressure for all particles,
//then apply a displacement based on that. There is an added if statement to handle surface tension for multiple weights of particles
void Cuda_solver::doubleDensityRelaxation(){
    if(h_params->totalParticles>0)
    {
        int threadsPerBlock = 256;
        int blocksPerGrid =
                (h_params->totalParticles + threadsPerBlock - 1) / threadsPerBlock;

        CudaDoubleDensityRelaxation<<<blocksPerGrid, threadsPerBlock>>>(d_particles,
                                                                        d_neightb_index, d_neightb_size, d_neightb_r, d_neightb_Lij);
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
    }
}

__global__ void CudaDoubleDensityRelaxation(Particle* particles,
                                            int* neightb_index, int* neightb_size, float* neightb_r, float* neightb_Lij)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i<params.totalParticles)
    {
        Particle& p = particles[i];

        float density = 0;
        float nearDensity = 0;

        for (int j = 0; j < neightb_size[i]; j++){
            const Particle& pNear = particles[neightb_index[GetIndexNeightb(i,j)]];// *neighbours[i].particles[j];

            float r = neightb_r[GetIndexNeightb(i,j)];
            float q = r / particleHeight;
            if (q>1) continue;
            float a = 1 - q;

            density += a*a * pNear.m * 20;
            nearDensity += a*a*a * pNear.m * 30;
        }
        p.pressure = params.stiffness * (density - p.m*restDensity);
        p.nearPressure = params.nearStiffness * nearDensity;
        float dx = 0, dy = 0;

        for (int j = 0; j < neightb_size[i]; j++){
            const Particle& pNear = particles[neightb_index[GetIndexNeightb(i,j)]];

            float diffX = pNear.posX - p.posX;
            float diffY = pNear.posY - p.posY;

            float r = neightb_r[GetIndexNeightb(i,j)];
            float q = r / particleHeight;
            if (q>1) continue;
            float a = 1 - q;
            float d = (deltaTime*deltaTime) * ((p.nearPressure + pNear.nearPressure)*a*a*a*53 + (p.pressure + pNear.pressure)*a*a*35) / 2;

            // weight is added to the denominator to reduce the change in dx based on its weight
            dx -= d * diffX / (r*p.m);
            dy -= d * diffY / (r*p.m);

            //surface tension is mapped with one type of particle,
            //this allows multiple weights of particles to behave appropriately
            if (p.m == pNear.m && multipleParticleTypes == true){
                dx += surfaceTension * diffX;
                dy += surfaceTension * diffY;
            }
        }

        p.posX += dx;
        p.posY += dy;

    }
}

void Cuda_solver::computeNextVelocity(){
    if(h_params->totalParticles>0)
    {
        int threadsPerBlock = 256;
        int blocksPerGrid =
                (h_params->totalParticles + threadsPerBlock - 1) / threadsPerBlock;

        CudaComputeNextVelocity<<<blocksPerGrid, threadsPerBlock>>>(d_particles, d_prevPos);
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
    }
}

__global__ void CudaComputeNextVelocity(Particle* particles, float2* prevPos)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i<params.totalParticles)
    {
        Particle& p = particles[i];
        p.velX = (p.posX - prevPos[i].x) / deltaTime;
        p.velY = (p.posY - prevPos[i].y) / deltaTime;

    }
}


//Only checks if particles have left window, and push them back if so
void Cuda_solver::resolveCollisions(){
    if(h_params->totalParticles>0)
    {
        int threadsPerBlock = 256;
        int blocksPerGrid =
                (h_params->totalParticles + threadsPerBlock - 1) / threadsPerBlock;

        CudaResolveCollisions<<<blocksPerGrid, threadsPerBlock>>>(d_particles, d_boundaries);
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
    }
}


__global__ void CudaResolveCollisions(Particle* particles, float3* boundaries)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i<params.totalParticles)
    {
        Particle& p = particles[i];

        for (int j = 0; j<4; j++){
            const float3& boundary = boundaries[j];
            float distance = boundary.x*p.posX + boundary.y*p.posY - boundary.z;

            if (distance < particleRadius){
                if (distance < 0)
                    distance = 0;
                p.velX += 0.1f*(particleRadius - distance) * boundary.x / deltaTime;
                p.velY += (particleRadius - distance) * boundary.y / deltaTime;
            }

            //The resolve collisions tends to overestimate the needed counter velocity, this limits that
            if (p.velX > 0.5) p.velX = 0.5;
            if (p.velY > 2) p.velY = 2;
            if (p.velX < -0.5) p.velX = -0.5;
            if (p.velY < -2) p.velY = -2;
        }

    }
}


//Iterates through every particle and multiplies its RGB values based on speed.
//speed^2 is just used to make the difference in speeds more noticeable.
void Cuda_solver::adjustColor()
{
    if(h_params->totalParticles>0)
    {
        int threadsPerBlock = 256;
        int blocksPerGrid =
                (h_params->totalParticles + threadsPerBlock - 1) / threadsPerBlock;

        CudaAdjustColor<<<blocksPerGrid, threadsPerBlock>>>(d_particles, d_savedParticleColors, d_particleTypes);
        hipDeviceSynchronize();
        getLastCudaError("Kernel execution failed");
    }
}


__global__ void CudaAdjustColor(Particle* particles, Vec4* savedParticleColors, ParticleType* particleTypes)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i<params.totalParticles)
    {
        const Particle& p = particles[i];
        const ParticleType& pt = particleTypes[i];

        float speed2 = p.temp/10.f;//.velX*p.velX + p.velY*p.velY;

        Vec4& color = savedParticleColors[i];
        color = pt.color;
        color.r *= 0.5f + velocityFactor*speed2;
        color.g *= 0.5f + velocityFactor*speed2;
        color.b *= 0.5f + velocityFactor*speed2;

    }
}
//Runs through all of the logic 7 times a frame
bool Cuda_solver::Update()
{

    checkCudaCall(hipMemcpyToSymbol(HIP_SYMBOL(params),h_params,sizeof(gpuParams)));
    //for (int step = 0; step<timeStep; step++)
    for (int step = 0; step<2; step++)
    {

        generateParticles();

        applyTemp();
        applyGravity();
        applyViscosity();
        advance();
        adjustSprings();
        updateMap();
        storeNeighbors();
        doubleDensityRelaxation();
        computeNextVelocity();
        resolveCollisions();

    }

    UpdateHostBuffers();
    //checkCudaCall(hipMemcpyFromSymbol(h_params,HIP_SYMBOL(params),sizeof(gpuParams)));

   // UpdateHostBuffers();
    /*
    ClearMap();

    for(int i=0; i<h_params->maxMapCoeff1;i++)
    {
        if(h_map_size[i]>0)
            printf(" %i", h_map_size[i]);
    }
*/
/*
    printf(" Start print \n");
    for(int ix=0; ix<h_params->mapWidth;ix++)
    {
        for(int iy=0; iy<h_params->mapHeight;iy++)
        {
            if(h_map_size[iy*mapWidth+ix]>0)
            {
                printf(" %i \n", h_map_size[iy*mapWidth+ix]);

                for(int ii=0; ii<h_map_size[iy*mapWidth+ix]; ii++)
                {
                     printf("ii %i ", h_map[ ii*h_params->maxMapCoeff1+iy*h_params->mapWidth+ix]);
                }
                printf("\n");
            }
        }
    }
    */
    return true;
}
